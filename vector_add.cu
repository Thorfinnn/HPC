
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel function to perform vector addition
__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int n = 100000; // Size of vectors
    int *a, *b, *c; // Host vectors
    int *d_a, *d_b, *d_c; // Device vectors
    int size = n * sizeof(int);

    // Allocate memory for host vectors
    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);

    // Initialize host vectors
    for (int i = 0; i < n; ++i) {
        a[i] = i;
        b[i] = i;
    }

    // Allocate memory for device vectors
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy host vectors to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    // Launch kernel
    vectorAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Verify the result
    for (int i = 100000-20; i < 100000; ++i) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(a);
    free(b);
    free(c);

    return 0;
}
